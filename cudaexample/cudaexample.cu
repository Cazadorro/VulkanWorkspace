
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

// function to add the elements of two arrays
__global__
void addConstant(float * matrix, unsigned width, unsigned height, float addend) {
    unsigned y_idx = (blockIdx.y * blockDim.y) + threadIdx.y;
    unsigned x_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    if(y_idx >= height || x_idx >= width){
        return;
    }
    unsigned unique_idx = (y_idx * width + x_idx);
    matrix[unique_idx] += addend;
}

int main(void){
    std::uint32_t width = 64;
    std::uint32_t height = 64;

    std::vector<float> host_matrix(width*height, 1.0);
    float* device_matrix;
    hipMalloc(&device_matrix, host_matrix.size() * sizeof(float));
    hipMemcpy(device_matrix, host_matrix.data(), host_matrix.size() * sizeof(float), hipMemcpyHostToDevice);

    dim3 block_size {32,32,1};
    dim3 grid_size{static_cast<std::uint32_t>(std::ceil(
            width / float(block_size.x))),
                   static_cast<std::uint32_t>(std::ceil(
                           height / float(block_size.y))),1};
    addConstant<<<grid_size, block_size>>>(device_matrix, 64, 64, 10.f);

    hipDeviceSynchronize();

    hipMemcpy(host_matrix.data(), device_matrix, host_matrix.size() * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(device_matrix);

    std::cout << "matrix size = " << host_matrix.size() << std::endl;
    for (const auto& value : host_matrix) {
        std::cout << value << std::endl;
    }
    return 0;
}